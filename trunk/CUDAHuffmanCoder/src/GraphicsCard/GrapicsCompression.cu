#include "hip/hip_runtime.h"
/*
 * GrapicsCompression.cpp
 *
 *  Created on: Sep 24, 2013
 *      Author: bjtal
 */

#include "GrapicsCompression.h"

void GPU::binVector(thrust::device_vector<float> & dev_vec, frequencyValues & map,
		double & time) {
	std::cout << "Using: " << omp_get_max_threads() << " threads\n";
	Timer::tic();

	thrust::sort(dev_vec.begin(), dev_vec.end());

	int num_bins = thrust::inner_product(dev_vec.begin(), dev_vec.end() - 1,
			dev_vec.begin() + 1, int(1), thrust::plus<int>(),
			thrust::not_equal_to<float>());

	thrust::device_vector<float> unique_device(num_bins);
	thrust::device_vector<int> counts_device(num_bins);

	thrust::reduce_by_key(dev_vec.begin(), dev_vec.end(),
			thrust::constant_iterator<int>(1), unique_device.begin(),
			counts_device.begin());
	std::cout << "Binning Time: " << Timer::toc() << std::endl;
	time += Timer::toc();

	Timer::tic();
	thrust::host_vector<float> unique_host = unique_device;
	thrust::host_vector<int> counts_host = counts_device;
	std::cout << "Unique Value and Count copy to HOST: " << Timer::toc()
			<< std::endl;
	time += Timer::toc();

	Timer::tic();
	for (int i = 0; i < num_bins; ++i)
		map[unique_host[i]] = new longValue(counts_host[i]);
	std::cout << "Conversion to HashMap: " << Timer::toc() << std::endl;
	time += Timer::toc();
}

void GPU::compress(const longValue & numberOfFloats) {
	thrust::host_vector<float> h_vec(numberOfFloats);

	srand(time(NULL));

	for (longValue i = 0; i < numberOfFloats; ++i)
		h_vec[i] = rand() % 9000;

	int numThreads = omp_get_max_threads();

	double totTime = 0;

	Timer::tic();
	HuffCode ** codes;
	codes = (HuffCode **) malloc(sizeof(HuffCode*) * numberOfFloats);
	frequencyValues map;
	Compressor comp(map);
	std::cout << "Pre Processing: " << Timer::toc() << std::endl;
	totTime += Timer::toc();

	// copy Data - Not adding to tot time for now
	Timer::tic();
	thrust::device_vector<float> dev_vec = h_vec;
	std::cout << "Copy Time (not-included): " << Timer::toc() << std::endl;
	//	time += Timer::toc();

	// bin data
	binVector(dev_vec, map, totTime);

	// generate the tree
	Timer::tic();
	comp.initialize();
	std::cout << "Tree creation: " << Timer::toc() << std::endl;
	totTime += Timer::toc();

	// CPU copy data from map to new values
	Timer::tic();
	longValue numProcess = ceil(numberOfFloats / (numThreads));
#pragma omp parallel for num_threads(numThreads)
	for (longValue i = 0; i < longValue((numThreads)); ++i) {

		longValue proc = min(numberOfFloats - (i * numProcess), numProcess);

		comp.compress(&h_vec[0] + (i * numProcess), codes + (i * numProcess),
				proc);
	}
	std::cout << "Swapping of values: " << Timer::toc() << std::endl;
	totTime += Timer::toc();

	std::cout << "Total Time: " << totTime << std::endl;
}
