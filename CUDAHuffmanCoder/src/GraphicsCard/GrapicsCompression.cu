#include "hip/hip_runtime.h"
/*
 * GrapicsCompression.cpp
 *
 *  Created on: Sep 24, 2013
 *      Author: bjtal
 */

#include "GrapicsCompression.h"

void GPU::binVector(thrust::device_vector<float> & dev_vec,
		frequencyValues & map, double & time) {
	Timer::tic();

	thrust::sort(dev_vec.begin(), dev_vec.end());

	int num_bins = thrust::inner_product(dev_vec.begin(), dev_vec.end() - 1,
			dev_vec.begin() + 1, int(1), thrust::plus<int>(),
			thrust::not_equal_to<float>());

	thrust::device_vector<float> unique_device(num_bins);
	thrust::device_vector<int> counts_device(num_bins);

	thrust::reduce_by_key(dev_vec.begin(), dev_vec.end(),
			thrust::constant_iterator<int>(1), unique_device.begin(),
			counts_device.begin());
	printf("Binning: %f\n", Timer::toc());
	time += Timer::toc();

	Timer::tic();
	thrust::host_vector<float> unique_host = unique_device;
	thrust::host_vector<int> counts_host = counts_device;
	printf("Device to host copy (counts & unique): %f\n", Timer::toc());
	time += Timer::toc();

	Timer::tic();
	for (int i = 0; i < num_bins; ++i)
		map[unique_host[i]] = new longValue(counts_host[i]);
	printf("Conversion to HashMap: %f\n", Timer::toc());
	time += Timer::toc();
}

void GPU::compress(const longValue & numberOfFloats) {
	thrust::host_vector<float> h_vec(numberOfFloats);

	srand(time(NULL));

	for (longValue i = 0; i < numberOfFloats; ++i)
		h_vec[i] = rand() % 9000;

	int numThreads = omp_get_max_threads();

	printf("\e[0;33mUsing: %d threads\n\e[0m", omp_get_max_threads());

	double totTime = 0;

	Timer::tic();
	HuffCode ** codes;
	codes = (HuffCode **) malloc(sizeof(HuffCode*) * numberOfFloats);
	frequencyValues map;
	Compressor comp(map);
	printf("Pre Processing: %f\n", Timer::toc());
	totTime += Timer::toc();

	// copy Data - Not adding to tot time for now
	Timer::tic();
	thrust::device_vector<float> dev_vec = h_vec;
	printf("\e[0;33mCopy Time (not-included): %f\n\e[0m", Timer::toc());
	//	time += Timer::toc();

	// bin data
	binVector(dev_vec, map, totTime);

	// generate the tree
	Timer::tic();
	comp.initialize();
	printf("Tree Creation: %f\n", Timer::toc());
	totTime += Timer::toc();

	// CPU copy data from map to new values
	Timer::tic();
	longValue numProcess = ceil(numberOfFloats / (numThreads));
#pragma omp parallel for num_threads(numThreads)
	for (longValue i = 0; i < longValue((numThreads)); ++i) {

		longValue proc = min(numberOfFloats - (i * numProcess), numProcess);

		comp.compress(&h_vec[0] + (i * numProcess), codes + (i * numProcess),
				proc);
	}
	printf("Swapping Values: %f\n", Timer::toc());
	totTime += Timer::toc();

	printf("\e[1;32mTotal Time: %f\n\e[0m", totTime);
}
