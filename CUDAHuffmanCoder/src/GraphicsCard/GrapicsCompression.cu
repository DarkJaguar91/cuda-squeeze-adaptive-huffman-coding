#include "hip/hip_runtime.h"
/*
 * GrapicsCompression.cpp
 *
 *  Created on: Sep 24, 2013
 *      Author: bjtal
 */

#include "GrapicsCompression.h"

__global__ void swapCodes(float * data, float * codeData, const int numData,
		const int numCodeData) {
	int pos = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x
			+ blockIdx.y * gridDim.x;

	if (pos < numData)
		for (int i = 0; i < numCodeData; ++i) {
//			printf("%f - %f\n", data[pos], codeData[i]);
			if (data[pos] == codeData[i]) {
				data[pos] = i;
				break;
			}
		}
}

void GPU::compress(const longValue & numberOfFloats) {
	srand(time(NULL));
	host_vec host(numberOfFloats);
	for (longValue i = 0; i < numberOfFloats; ++i) {
		host[i] = rand() % 9000;
	}
	frequencyValues map;
	std::vector<unsigned char> codes;
	double totTime = 0;
	compressData(host, map, codes, totTime);
}

void GPU::compress(char * inputFileName, char * outputFileName) {
	Timer::tic();
	FILE * in;
	in = fopen(inputFileName, "r");
	longValue numFloats;
	size_t res = fread(&numFloats, sizeof(longValue), 1, in);
	host_vec host;
	host.resize(numFloats);
	res = fread(&host[0], sizeof(float), numFloats, in);
	fclose(in);
	double readTime = Timer::toc();
	printf("Time to read data: %f\n", readTime);

	std::vector<unsigned char> charArray;
	frequencyValues map;
	double processTime = 0;
	compressData(host, map, charArray, processTime);

	Timer::tic();
	FILE * out;
	out = fopen(outputFileName, "w");

	longValue numUnique = map.size();
	fwrite(&numUnique, sizeof(longValue), 1, out);

	for (frequencyValues::iterator it = map.begin(); it != map.end(); ++it){
		float uval = it->first;
		longValue cnt = *it->second;

		fwrite(&uval, sizeof(float), 1, out);
		fwrite(&cnt, sizeof(longValue), 1, out);
	}

	longValue numChars = charArray.size();
	fwrite(&numChars, sizeof(longValue), 1, out);
	fwrite(&charArray[0], sizeof(unsigned char), numChars, out);

	fclose(out);
	double writeTime = Timer::toc();
	printf("Time to write data: %f\n", writeTime);

	printf("Total Time: %f\n", writeTime + readTime + processTime);
}

void GPU::compressData(host_vec & host, frequencyValues & map, std::vector<unsigned char> & charCodes, double & totTime){
	int numThreads = omp_get_max_threads();
	longValue numberOfFloats = host.size();

	printf("Using: %d threads\n", omp_get_max_threads());

	Timer::tic();
	HuffCode ** codes;
	codes = (HuffCode **) malloc(sizeof(HuffCode*) * numberOfFloats);
	Compressor comp(map);
	printf("Pre Processing: %f\n", Timer::toc());
	totTime += Timer::toc();

	// copy Data - Not adding to tot time for now
	Timer::tic();
	dev_vec dev = host;
	printf("Copy Time (included): %f\n", Timer::toc());
	totTime += Timer::toc();

	// sort data
	Timer::tic();
	thrust::sort(dev.begin(), dev.end());
	printf("sort: %f\n", Timer::toc());
	totTime += Timer::toc();

	// bin data
	Timer::tic();
	int num_bins = thrust::inner_product(dev.begin(), dev.end() - 1,
			dev.begin() + 1, int(1), thrust::plus<int>(),
			thrust::not_equal_to<float>());

	dev_vec unique_device(num_bins);
	dev_vec_int counts_device(num_bins);

	thrust::reduce_by_key(dev.begin(), dev.end(),
			thrust::constant_iterator<int>(1), unique_device.begin(),
			counts_device.begin());
	printf("Binning: %f\n", Timer::toc());
	totTime += Timer::toc();

	Timer::tic();
	host_vec unique_host = unique_device;
	host_vec_int counts_host = counts_device;
	printf("Device to host copy (counts & unique): %f\n", Timer::toc());
	totTime += Timer::toc();

	Timer::tic();
	for (int i = 0; i < num_bins; ++i)
		map[unique_host[i]] = new longValue(counts_host[i]);
	// some memory management Remove counts as we dont need them
	counts_device.clear();
	counts_device.shrink_to_fit();
	counts_host.clear();
	counts_host.shrink_to_fit();
	unique_host.clear(); // clear this as its duplicated
	unique_host.shrink_to_fit();

	printf("Conversion to HashMap: %f\n", Timer::toc());
	totTime += Timer::toc();

	// generate the tree
	Timer::tic();
	comp.initialize();
	printf("Tree Creation: %f\n", Timer::toc());
	totTime += Timer::toc();

	//	// swap data GPU -->
	//	Timer::tic();
	//	dev = host;
	////	thrust::host_vector<int> length(num_bins);
	////	thrust::host_vector<int> startPos(num_bins);
	////	thrust::host_vector<bool> codeArr(comp.getTotalCodeLength());
	////	int pos = 0;
	////	for (int i = 0; i < num_bins; ++i) {
	////		HuffCode code = *(comp.map[unique_host[i]]);
	////		length[i] = code.size();
	////		startPos[i] = pos;
	////		pos += code.size();
	////
	////		for (int z = 0; z < code.size(); ++z)
	////			codeArr[startPos[i] + z] = code[z];
	////	}
	//	printf("Create array times: %f\n", Timer::toc());
	//
	//	Timer::tic();
	//	dim3 grid, block;
	//	block.x = 16;
	//	block.y = 16;
	//	block.z = 1;
	//
	//	grid.x = ceil(sqrt((numberOfFloats / (block.x * block.y * block.z))));
	//	grid.y = grid.x;
	//	grid.z = 1;
	//	float * data_ptr = thrust::raw_pointer_cast(dev.data());
	//	float * codeData_ptr = thrust::raw_pointer_cast(unique_device.data());
	//	swapCodes<<<grid, block>>>(data_ptr, codeData_ptr, numberOfFloats, num_bins);
	//	hipDeviceSynchronize();
	//	printf("Swap Time: %f\n", Timer::toc());

	// CPU copy data from map to new values
	Timer::tic();
	longValue numProcess = ceil(numberOfFloats / (numThreads));
#pragma omp parallel for num_threads(numThreads)
	for (longValue i = 0; i < longValue((numThreads)); ++i) {
		longValue proc = min(numberOfFloats - (i * numProcess), numProcess);

		comp.compress(&host[0] + (i * numProcess), codes + (i * numProcess),
				proc);
	}
	printf("Swapping Values: %f\n", Timer::toc());
	totTime += Timer::toc();

	printf("Process before char creation: %f\n", totTime);

	Timer::tic();
	///////////////////////////
	longValue cnt = 0;
	double size = 0;

	unsigned char b = 0;
	for (longValue i = 0; i < numberOfFloats; ++i) {
		for (int z = 0; z < codes[i]->size(); ++z) {
			if (codes[i]->at(z) == 1)
				b |= (1 << (7 - (cnt % 8)));
			else
				b &= ~(1 << (7 - (cnt % 8)));
			cnt++;
			if (cnt % 8 == 0)
			{
				charCodes.push_back(b);
				b = 0;
			}
			size++;
		}
	}
	///////////////////////////////////////
	///////////////////////////////////////
//	longValue cnt = 0;
//	for (longValue i = 0; i < numberOfFloats; ++i) {
//		cnt += codes[i]->size();
//	}
//
//	double size = cnt;
//
//	HuffCode array(cnt);
//
//
//	cnt = 0;
//	for (longValue i = 0; i < numberOfFloats; ++i) {
//		for (int z = 0; z < codes[i]->size(); ++z) {
//			array[cnt] = codes[i]->at(z);
//			cnt++;
//		}
//	}
//	printf("Flattening array: %f\n", Timer::toc());
//	totTime += Timer::toc();
//
//	Timer::tic();
//	int numChars = int(ceil(array.size() / float(8)));
//	charCodes.resize(numChars);
//	int i = 0;
//#pragma omp parallel for private(i) schedule(dynamic) shared(array)
//	for (i = 0; i < numChars; ++i) {
//		unsigned char b = 0;
//		for (int z = 0; z < 8; ++z) {
//			if (array[i * 8 + z] == 1)
//				b |= (1 << (7 - (z % 8)));
//			else
//				b &= ~(1 << (7 - (z % 8)));
//		}
//		charCodes[i] = b;
//	}
	////////////////////////////////////////////////////////////////////
	printf("Converting to char: %f\n", Timer::toc());
	totTime += Timer::toc();

	printf("Total Time for processing: %f\n", totTime);

	size = ceil(size / 8.0f) * sizeof(unsigned char);
	size += map.size() * sizeof(float) + map.size() * sizeof(longValue) + sizeof(longValue) * 2;
	double inputSize = float(numberOfFloats * sizeof(float) + sizeof(longValue));
	printf("Size of output file: %f\n", size / 1024.0f / 1024.0f);
	printf("Size of input file: %f\n", inputSize / 1024.0f / 1024.0f);
	printf("compression ratio: %f\n", size / inputSize);

	delete [] codes;

	/// check
	Timer::tic();
	Decompressor decomp(map);
	decomp.initialize();
	std::vector<float> floats;

	HuffCode array;
	for (int i = 0; i < charCodes.size(); ++i){
		unsigned char c = charCodes.at(i);
		for (long i = 0; i < 8; ++i) {
			array.push_back(((c >> (7 - i)) & 1));
		}
	}
	decomp.decode(array, floats);
	printf("Timer to decompress: %f\n", Timer::toc());

	for (int i = 0; i < floats.size(); ++i)
		if (floats[i] != host[i])
			printf("failed %d->   %f   <>   %f\n", i, floats[i], host[i]);
}
