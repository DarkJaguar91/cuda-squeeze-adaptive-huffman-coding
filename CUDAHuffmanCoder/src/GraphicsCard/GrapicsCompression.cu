#include "hip/hip_runtime.h"
/*
 * GrapicsCompression.cpp
 *
 *  Created on: Sep 24, 2013
 *      Author: bjtal
 */

#include "GrapicsCompression.h"

__global__ void swapCodes(float * data, float * codeData, const int numData,
		const int numCodeData) {
	int pos = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x
			+ blockIdx.y * gridDim.x;

	if (pos < numData)
		for (int i = 0; i < numCodeData; ++i) {
//			printf("%f - %f\n", data[pos], codeData[i]);
			if (data[pos] == codeData[i]) {
				data[pos] = i;
				break;
			}
		}
}

void GPU::compress(const longValue & numberOfFloats) {
	srand(time(NULL));
	host_vec host(numberOfFloats);
	for (longValue i = 0; i < numberOfFloats; ++i) {
		host[i] = rand() % 9000;
	}

	int numThreads = omp_get_max_threads();

	printf("\e[0;33mUsing: %d threads\n\e[0m", omp_get_max_threads());

	double totTime = 0;

	Timer::tic();
	HuffCode ** codes;
	codes = (HuffCode **) malloc(sizeof(HuffCode*) * numberOfFloats);
	frequencyValues map;
	Compressor comp(map);
	printf("Pre Processing: %f\n", Timer::toc());
	totTime += Timer::toc();

	// copy Data - Not adding to tot time for now
	Timer::tic();
	dev_vec dev = host;
	printf("\e[0;33mCopy Time (not-included): %f\n\e[0m", Timer::toc());
	//	time += Timer::toc();

	// sort data
	Timer::tic();
	thrust::sort(dev.begin(), dev.end());
	printf("sort: %f\n", Timer::toc());
	totTime += Timer::toc();

	// bin data
	Timer::tic();
	int num_bins = thrust::inner_product(dev.begin(), dev.end() - 1,
			dev.begin() + 1, int(1), thrust::plus<int>(),
			thrust::not_equal_to<float>());

	dev_vec unique_device(num_bins);
	dev_vec_int counts_device(num_bins);

	thrust::reduce_by_key(dev.begin(), dev.end(),
			thrust::constant_iterator<int>(1), unique_device.begin(),
			counts_device.begin());
	printf("Binning: %f\n", Timer::toc());
	totTime += Timer::toc();

	Timer::tic();
	host_vec unique_host = unique_device;
	host_vec_int counts_host = counts_device;
	printf("Device to host copy (counts & unique): %f\n", Timer::toc());
	totTime += Timer::toc();

	Timer::tic();
	for (int i = 0; i < num_bins; ++i)
		map[unique_host[i]] = new longValue(counts_host[i]);
	// some memory management Remove counts as we dont need them
	counts_device.clear();
	counts_device.shrink_to_fit();
	counts_host.clear();
	counts_host.shrink_to_fit();
	unique_host.clear(); // clear this as its duplicated
	unique_host.shrink_to_fit();

	printf("Conversion to HashMap: %f\n", Timer::toc());
	totTime += Timer::toc();

	// generate the tree
	Timer::tic();
	comp.initialize();
	printf("Tree Creation: %f\n", Timer::toc());
	totTime += Timer::toc();

//	// swap data GPU -->
//	Timer::tic();
//	dev = host;
////	thrust::host_vector<int> length(num_bins);
////	thrust::host_vector<int> startPos(num_bins);
////	thrust::host_vector<bool> codeArr(comp.getTotalCodeLength());
////	int pos = 0;
////	for (int i = 0; i < num_bins; ++i) {
////		HuffCode code = *(comp.map[unique_host[i]]);
////		length[i] = code.size();
////		startPos[i] = pos;
////		pos += code.size();
////
////		for (int z = 0; z < code.size(); ++z)
////			codeArr[startPos[i] + z] = code[z];
////	}
//	printf("Create array times: %f\n", Timer::toc());
//
//	Timer::tic();
//	dim3 grid, block;
//	block.x = 16;
//	block.y = 16;
//	block.z = 1;
//
//	grid.x = ceil(sqrt((numberOfFloats / (block.x * block.y * block.z))));
//	grid.y = grid.x;
//	grid.z = 1;
//	float * data_ptr = thrust::raw_pointer_cast(dev.data());
//	float * codeData_ptr = thrust::raw_pointer_cast(unique_device.data());
//	swapCodes<<<grid, block>>>(data_ptr, codeData_ptr, numberOfFloats, num_bins);
//	hipDeviceSynchronize();
//	printf("Swap Time: %f\n", Timer::toc());

	// CPU copy data from map to new values
	Timer::tic();
	longValue numProcess = ceil(numberOfFloats / (numThreads));
#pragma omp parallel for num_threads(numThreads)
	for (longValue i = 0; i < longValue((numThreads)); ++i) {

		longValue proc = min(numberOfFloats - (i * numProcess), numProcess);

		comp.compress(&host[0] + (i * numProcess), codes + (i * numProcess),
				proc);
	}
	printf("Swapping Values: %f\n", Timer::toc());
	totTime += Timer::toc();

	printf("\e[1;32mTotal Time: %f\n\e[0m", totTime);
}
